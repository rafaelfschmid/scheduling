#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : sorting_segments.cu
 Author      : Rafael Schmid
 Version     :
 Copyright   : Your copyright notice
 Description : Compute sum of reciprocals using STL on CPU and Thrust on GPU
 ============================================================================

 COMPILAR USANDO O SEGUINTE COMANDO:

 nvcc segmented_sort.cu -o segmented_sort -std=c++11 --expt-extended-lambda -I"/home/schmid/Dropbox/Unicamp/workspace/sorting_segments/moderngpu-master/src"

 */

#include <moderngpu/kernel_segsort.hxx>

#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include <hip/hip_runtime.h>

#ifndef ELAPSED_TIME
#define ELAPSED_TIME 0
#endif

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 32
#endif

void cudaTest(hipError_t error) {
	if (error != hipSuccess) {
		printf("cuda returned error %s (code %d), line(%d)\n",
				hipGetErrorString(error), error, __LINE__);
		exit (EXIT_FAILURE);
	}
}

template<typename T>
void print(T* vec, uint t, uint m) {
	std::cout << "\n";
	for (uint i = 0; i < t; i++) {
		for (uint j = 0; j < m; j++) {
			std::cout << vec[i * m + j] << " ";
		}
		std::cout << "\n";
	}

}

template<typename T>
void print(T* vec, uint t) {
	std::cout << "\n";
	for (uint i = 0; i < t; i++) {
		std::cout << vec[i] << " ";
	}
	std::cout << "\n";
}

__global__ void min_min(float* machines, float* completion_times, bool* task_map, bool* task_deleted,
		float* d_reduc_comp, int* d_reduc_ind_t, int* d_reduc_ind_m,
			int m, int t, float MAX_FLOAT) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int tId = threadIdx.x;
	extern __shared__ int vec[];

	float *s_comp_times = (float*)&vec[0];
	int *s_ind_t = (int*)&vec[BLOCK_SIZE];
	int *s_ind_m = (int*)&vec[2*BLOCK_SIZE];

	uint min = 0;
	uint imin = 0;
	uint jmin = 0;
	float min_value = MAX_FLOAT;

	if(!task_deleted[i]) {
		for (int j = 0; j < m; j++) {

			if (completion_times[j] + machines[j * t + i] < min_value) {
				imin = i;
				jmin = j;
				min = jmin * t + imin;
				min_value = completion_times[jmin] + machines[min];
			}
		}
	}

	s_comp_times[tId] = min_value;
	s_ind_t[tId] = imin;
	s_ind_m[tId] = jmin;

	__syncthreads();

	for(int e = BLOCK_SIZE/2; e > 0; e/=2)
	{
		if (tId < e) {
			if ((s_comp_times[tId + e] < s_comp_times[tId])
					|| (s_comp_times[tId + e] == s_comp_times[tId]
							&& s_ind_t[tId + e] < s_ind_t[tId])) {
				s_comp_times[tId] = s_comp_times[tId + e];
				s_ind_t[tId] = s_ind_t[tId + e];
				s_ind_m[tId] = s_ind_m[tId + e];
			}
		}
		__syncthreads();
	}

	if(tId == 0) {
		d_reduc_comp [blockIdx.x] = s_comp_times[0];
		d_reduc_ind_t[blockIdx.x] = s_ind_t[0];
		d_reduc_ind_m[blockIdx.x] = s_ind_m[0];
	}
}

__global__ void reduction(float* d_reduc_comp, int* d_reduc_ind_t, int* d_reduc_ind_m) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int tId = threadIdx.x;

	extern __shared__ int vec[];
	float *s_comp_times = (float*)&vec[0];
	int *s_ind_t = (int*)&vec[BLOCK_SIZE];
	int *s_ind_m = (int*)&vec[2*BLOCK_SIZE];

	s_comp_times[tId] = d_reduc_comp[i];
	s_ind_t[tId] = d_reduc_ind_t[i];
	s_ind_m[tId] = d_reduc_ind_m[i];

	__syncthreads();

	for(int e = BLOCK_SIZE/2; e > 0; e/=2)
	{
		if (tId < e) {
			if ((s_comp_times[tId + e] < s_comp_times[tId])
					|| (s_comp_times[tId + e] == s_comp_times[tId]
							&& s_ind_t[tId + e] < s_ind_t[tId])) {
				s_comp_times[tId] = s_comp_times[tId + e];
				s_ind_t[tId] = s_ind_t[tId + e];
				s_ind_m[tId] = s_ind_m[tId + e];
			}
		}
		__syncthreads();
	}

	if(tId == 0) {
		d_reduc_comp [blockIdx.x] = s_comp_times[0];
		d_reduc_ind_t[blockIdx.x] = s_ind_t[0];
		d_reduc_ind_m[blockIdx.x] = s_ind_m[0];
	}
}

__global__ void block_reduction(float* completion_times, bool* task_map, bool* task_deleted,
		float* d_reduc_comp, int* d_reduc_ind_t, int* d_reduc_ind_m, int t) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int tId = threadIdx.x;

	extern __shared__ int vec[];
	float *s_comp_times = (float*)&vec[0];
	int *s_ind_t = (int*)&vec[blockDim.x];
	int *s_ind_m = (int*)&vec[2*blockDim.x];

	s_comp_times[tId] = d_reduc_comp[i];
	s_ind_t[tId] = d_reduc_ind_t[i];
	s_ind_m[tId] = d_reduc_ind_m[i];

	__syncthreads();

	for(int e = blockDim.x/2; e > 0; e/=2)
	{
		if (tId < e) {
			if ((s_comp_times[tId + e] < s_comp_times[tId])
					|| (s_comp_times[tId + e] == s_comp_times[tId]
							&& s_ind_t[tId + e] < s_ind_t[tId])) {
				s_comp_times[tId] = s_comp_times[tId + e];
				s_ind_t[tId] = s_ind_t[tId + e];
				s_ind_m[tId] = s_ind_m[tId + e];
			}
		}
		__syncthreads();
	}

	if(tId == 0) {
		task_deleted[ s_ind_t[0] ] = true;
		task_map[ s_ind_m[0] * t + s_ind_t[0] ] = true;
		completion_times[ s_ind_m[0] ] = s_comp_times[0];
	}
}

int main(int argc, char** argv) {

	int t, m;

	if (argc < 3) {
			printf("Parameters missing: <number of tasks> <number of machines>\n\n");
			return 0;
	}

	t = atoi(argv[1]);
	m = atoi(argv[2]);

	uint mem_size_machines 			= sizeof(float) * (m * t);
	uint mem_size_completion_times 	= sizeof(float) * (m);
	uint mem_size_task_deleted 		= sizeof(bool) * (t);
	uint mem_size_task_map 			= sizeof(bool) * (m * t);
	uint mem_size_reduc_comp		= sizeof(float) * (t/BLOCK_SIZE);
	uint mem_size_reduc_ind			= sizeof(int) * (t/BLOCK_SIZE);

	float *machines				= (float *) malloc(mem_size_machines);
	float *completion_times 	= (float *) malloc(mem_size_completion_times);
	bool *task_deleted			= (bool  *) malloc(mem_size_task_deleted);
	bool *task_map 				= (bool  *) malloc(mem_size_task_map);

	float aux;
	for (int i = 0; i < t; i++) {
		for (int j = 0; j < m; j++) {
			int a = scanf("%f", &aux);

			machines[j * t + i] = aux;
			task_map[j * t + i] = false;
			completion_times[j] = 0;
		}
		task_deleted[i] = false;
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	float *d_machines, *d_completion_times, *d_reduc_comp;
	int *d_reduc_ind_t, *d_reduc_ind_m;
	bool *d_task_deleted, *d_task_map;
	float MAX_FLOAT = std::numeric_limits<float>::max();

	cudaTest(hipMalloc((void **) &d_machines, mem_size_machines));
	cudaTest(hipMalloc((void **) &d_completion_times, mem_size_completion_times));
	cudaTest(hipMalloc((void **) &d_task_deleted, mem_size_task_deleted));
	cudaTest(hipMalloc((void **) &d_task_map, mem_size_task_map));

	cudaTest(hipMalloc((void **) &d_reduc_comp, mem_size_reduc_comp));
	cudaTest(hipMalloc((void **) &d_reduc_ind_t, mem_size_reduc_ind));
	cudaTest(hipMalloc((void **) &d_reduc_ind_m, mem_size_reduc_ind));

	// copy host memory to device
	cudaTest(hipMemcpy(d_machines, machines, mem_size_machines, hipMemcpyHostToDevice));
	cudaTest(hipMemcpy(d_completion_times, completion_times, mem_size_completion_times, hipMemcpyHostToDevice));
	cudaTest(hipMemcpy(d_task_deleted, task_deleted, mem_size_task_deleted, hipMemcpyHostToDevice));
	cudaTest(hipMemcpy(d_task_map, task_map, mem_size_task_map, hipMemcpyHostToDevice));


	hipEventRecord(start);
	for(int k = 0; k < t; k++) {
		dim3 dimBlock(BLOCK_SIZE);
		int dim = t/BLOCK_SIZE;
		dim3 dimGrid(dim);
		min_min<<<dimGrid, dimBlock, BLOCK_SIZE * sizeof(float) +
				BLOCK_SIZE * sizeof(int) + BLOCK_SIZE * sizeof(int) >>>
				(d_machines, d_completion_times, d_task_map, d_task_deleted,
						d_reduc_comp, d_reduc_ind_t, d_reduc_ind_m, m, t, MAX_FLOAT);

		for( ; dim > BLOCK_SIZE; dim/=BLOCK_SIZE) {
			dim3 block(BLOCK_SIZE);
			dim3 grid_b(dim/BLOCK_SIZE);
			reduction<<<grid_b, block, BLOCK_SIZE * sizeof(float) +
					BLOCK_SIZE * sizeof(int) + BLOCK_SIZE * sizeof(int) >>>
				(d_reduc_comp, d_reduc_ind_t, d_reduc_ind_m);
		}
		dim3 block(dim);
		dim3 grid_b(1);
		block_reduction<<<grid_b, block, dim * sizeof(float) + dim * sizeof(int) +
				dim * sizeof(int) >>> (d_completion_times, d_task_map, d_task_deleted,
				d_reduc_comp, d_reduc_ind_t, d_reduc_ind_m, t);
	}
	hipEventRecord(stop);

	hipError_t errSync = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess)
		printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
	if (errAsync != hipSuccess)
		printf("Async kernel error: %s\n", hipGetErrorString(errAsync));

	if (ELAPSED_TIME == 1) {
		hipEventSynchronize(stop);
		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);
		std::cout << milliseconds << "\n";
	}

	hipDeviceSynchronize();

	cudaTest(hipMemcpy(completion_times, d_completion_times, mem_size_completion_times, hipMemcpyDeviceToHost));
	cudaTest(hipMemcpy(task_map, d_task_map, mem_size_task_map, hipMemcpyDeviceToHost));

	hipFree(d_machines);
	hipFree(d_completion_times);
	hipFree(d_task_map);
	hipFree(d_task_deleted);

	hipFree(d_reduc_comp);
	hipFree(d_reduc_ind_t);
	hipFree(d_reduc_ind_m);

	if (ELAPSED_TIME != 1) {
		//print(machines, m, t);
		//print(task_index, m, t);
		print(completion_times, m);
	}

	free(task_deleted);
	free(task_map);
	free(machines);
	free(completion_times);

	return 0;
}
