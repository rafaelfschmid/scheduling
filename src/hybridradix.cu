#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : sorting_segments.cu
 Author      : Rafael Schmid
 Version     :
 Copyright   : Your copyright notice
 Description : Compute sum of reciprocals using STL on CPU and Thrust on GPU
 ============================================================================

 COMPILAR USANDO O SEGUINTE COMANDO:

 nvcc segmented_sort.cu -o segmented_sort -std=c++11 --expt-extended-lambda -I"/home/schmid/Dropbox/Unicamp/workspace/sorting_segments/moderngpu-master/src"

 */

#include <cub/util_allocator.cuh>
#include <cub/device/device_segmented_radix_sort.cuh>

#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <utility>
#include <iostream>
#include <bitset>
#include <math.h>
#include <time.h>
#include <chrono>
#include <hip/hip_runtime.h>
//#include <cstdlib>
#include <iostream>
#include <chrono>

#ifndef ELAPSED_TIME
#define ELAPSED_TIME 0
#endif

#ifndef EXECUTIONS
#define EXECUTIONS 11
#endif


using namespace std::chrono;
using namespace hipcub;

void cudaTest(hipError_t error) {
	if (error != hipSuccess) {
		printf("cuda returned error %s (code %d), line(%d)\n",
				hipGetErrorString(error), error, __LINE__);
		exit (EXIT_FAILURE);
	}
}

template<typename T>
void print(T* vec, uint t, uint m) {
	std::cout << "\n";
	for (uint i = 0; i < t; i++) {
		for (uint j = 0; j < m; j++) {
			std::cout << vec[i * m + j] << " ";
		}
		std::cout << "\n";
	}

}

template<typename T>
void print(T* vec, uint t) {
	std::cout << "\n";
	for (uint i = 0; i < t; i++) {
		std::cout << vec[i] << " ";
	}
	std::cout << "\n";


}

void printSeg(int* host_data, uint num_seg, uint num_ele) {
	std::cout << "\n";
	for (uint i = 0; i < num_seg; i++) {
		std::cout << host_data[i] << " ";
	}
	std::cout << num_ele << " ";
	std::cout << "\n";
}

void min_min_sorted(float* machines, uint* task_index, float* completion_times, bool* task_map,
		bool* task_deleted, uint* machine_current_index, int m, int t) {

	uint min = 0;
	uint imin = 0;
	float min_value;

	for(int k = 0; k < t; k++) {

		min_value = std::numeric_limits<float>::max();

		for (int i = 0; i < m; i++) {

			int j = machine_current_index[i];
			while (task_deleted[task_index[i * t + j]]) {
				j++;
			}
			machine_current_index[i] = j;

			if (completion_times[i] + machines[i * t + j] < min_value) {
				min = i * t + j;
				imin = i;
				min_value = completion_times[imin] + machines[min];
			}
		}
		task_deleted[task_index[min]] = true;
		task_map[task_index[min] * m + imin] = true;
		completion_times[imin] = min_value;
	}

}

int main(int argc, char** argv) {

	int t, m;

	if (argc < 3) {
			printf("Parameters missing: <number of tasks> <number of machines>\n\n");
			return 0;
	}

	t = atoi(argv[1]);
	m = atoi(argv[2]);

	uint mem_size_seg = sizeof(int) * (m + 1);
	uint mem_size_machines = sizeof(float) * (m * t);
	uint mem_size_task_index = sizeof(uint) * (m * t);

	bool *task_deleted = (bool *) malloc(sizeof(bool) * t);
	bool *task_map = (bool *) malloc(sizeof(bool) * (t * m));

	int *segments = (int *) malloc(mem_size_seg);
	float *machines = (float *) malloc(mem_size_machines);
	uint *task_index = (uint *) malloc(mem_size_task_index);

	float *completion_times = (float *) malloc(sizeof(float) * (m));
	uint *machine_current_index = (uint *) malloc(sizeof(uint) * (m));

	float aux;
	for (int i = 0; i < t; i++) {
		for (int j = 0; j < m; j++) {
			int a = scanf("%f", &aux);

			task_index[j * t + i] = i;
			machines[j * t + i] = aux;
			segments[j] = j*t;

			task_map[i * m + j] = false;
			completion_times[j] = 0;
			machine_current_index[j] = 0;
		}
		task_deleted[i] = false;
	}
	segments[m] = m*t;

	//print(machines, m, t);
	//print(task_index, m, t);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	uint *d_task_index, *d_task_index_out;
	int *d_segments;
	float *d_machines, *d_machines_out;
	void *d_temp = NULL;
	size_t temp_bytes = 0;

	cudaTest(hipMalloc((void **) &d_segments, mem_size_seg));
	cudaTest(hipMalloc((void **) &d_machines, mem_size_machines));
	cudaTest(hipMalloc((void **) &d_machines_out, mem_size_machines));
	cudaTest(hipMalloc((void **) &d_task_index, mem_size_task_index));
	cudaTest(hipMalloc((void **) &d_task_index_out, mem_size_task_index));

	hipEventRecord(start);
	// copy host memory to device
	cudaTest(hipMemcpy(d_segments, segments, mem_size_seg, hipMemcpyHostToDevice));
	cudaTest(hipMemcpy(d_machines, machines, mem_size_machines, hipMemcpyHostToDevice));
	cudaTest(hipMemcpy(d_task_index, task_index, mem_size_task_index, hipMemcpyHostToDevice));

	hipcub::DeviceSegmentedRadixSort::SortPairs(d_temp, temp_bytes, d_task_index, d_task_index,
			d_task_index, d_task_index_out, m * t,	m, d_segments, d_segments + 1);
	cudaTest(hipMalloc((void **) &d_temp, temp_bytes));
	hipcub::DeviceSegmentedRadixSort::SortPairs(d_temp, temp_bytes, d_machines, d_machines_out,
			d_task_index, d_task_index_out, m * t,	m, d_segments, d_segments + 1);

	cudaTest(hipMemcpy(task_index, d_task_index_out, mem_size_task_index, hipMemcpyDeviceToHost));
	cudaTest(hipMemcpy(machines, d_machines_out, mem_size_machines, hipMemcpyDeviceToHost));

	min_min_sorted(machines, task_index, completion_times, task_map, task_deleted, machine_current_index, m, t);

	hipEventRecord(stop);

	hipError_t errSync = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess)
		printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
	if (errAsync != hipSuccess)
		printf("Async kernel error: %s\n", hipGetErrorString(errAsync));

	if (ELAPSED_TIME == 1) {
		hipEventSynchronize(stop);
		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);
		std::cout << milliseconds << "\n";
	}

	hipFree(d_segments);
	hipFree(d_machines);
	hipFree(d_machines_out);
	hipFree(d_task_index);
	hipFree(d_task_index_out);
	hipFree(d_temp);

	if (ELAPSED_TIME != 1) {
		//print(machines, m, t);
		//print(task_index, m, t);
		print(completion_times, m);
	}

	free(task_deleted);
	free(task_map);
	free(machines);
	free(task_index);
	free(segments);
	free(completion_times);
	free(machine_current_index);

	return 0;
}
