#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : sorting_segments.cu
 Author      : Rafael Schmid
 Version     :
 Copyright   : Your copyright notice
 Description : Compute sum of reciprocals using STL on CPU and Thrust on GPU
 ============================================================================

 COMPILAR USANDO O SEGUINTE COMANDO:

 nvcc segmented_sort.cu -o segmented_sort -std=c++11 --expt-extended-lambda -I"/home/schmid/Dropbox/Unicamp/workspace/sorting_segments/moderngpu-master/src"

 */

#include <cub/util_allocator.cuh>
#include <cub/device/device_segmented_radix_sort.cuh>

#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include <hip/hip_runtime.h>

#ifndef ELAPSED_TIME
#define ELAPSED_TIME 0
#endif

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 32
#endif

void cudaTest(hipError_t error) {
	if (error != hipSuccess) {
		printf("cuda returned error %s (code %d), line(%d)\n",
				hipGetErrorString(error), error, __LINE__);
		exit (EXIT_FAILURE);
	}
}

template<typename T>
void print(T* vec, uint t, uint m) {
	std::cout << "\n";
	for (uint i = 0; i < t; i++) {
		for (uint j = 0; j < m; j++) {
			std::cout << vec[i * m + j] << " ";
		}
		std::cout << "\n";
	}

}

template<typename T>
void print(T* vec, uint t) {
	std::cout << "\n";
	for (uint i = 0; i < t; i++) {
		std::cout << vec[i] << " ";
	}
	std::cout << "\n";


}

__global__ void transpose(const float *machines, float *machines_out,
		const uint *task_index, uint* task_index_out, int t, int m)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
  	int col = blockIdx.x * blockDim.x + threadIdx.x;

  	//for (int e = 0; e < t; e += BLOCK_SIZE)
  	machines_out[col * t + row] = machines[row * m + col];
  	task_index_out[col * t + row] = task_index[row * m + col];

}

int main(int argc, char** argv) {

	int t, m;

	/*if (argc < 3) {
			printf("Parameters missing: <number of tasks> <number of machines>\n\n");
			return 0;
	}
	t = atoi(argv[1]);
	m = atoi(argv[2]);
	*/
	int a = scanf("%d", &t);
	int a = scanf("%d", &m);

	uint mem_size_machines 			= sizeof(float) * (m * t);
	uint mem_size_task_index		= sizeof(uint) * (m * t);

	float *machines				= (float *) malloc(mem_size_machines);
	uint *task_index 			= (uint  *) malloc(mem_size_task_index);

	float aux;
	for (int i = 0; i < t; i++) {
		for (int j = 0; j < m; j++) {
			int a = scanf("%f", &aux);

			task_index[i * m + j] = j;
			machines[i * m + j] = aux;
		}
	}

	print(machines, t, m);
	print(task_index, t, m);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	uint *d_task_index, *d_task_index_out;
	float *d_machines, *d_machines_out;

	cudaTest(hipMalloc((void **) &d_machines, mem_size_machines));
	cudaTest(hipMalloc((void **) &d_machines_out, mem_size_machines));
	cudaTest(hipMalloc((void **) &d_task_index, mem_size_task_index));
	cudaTest(hipMalloc((void **) &d_task_index_out, mem_size_task_index));

	// copy host memory to device
	cudaTest(hipMemcpy(d_machines, machines, mem_size_machines, hipMemcpyHostToDevice));
	cudaTest(hipMemcpy(d_task_index, task_index, mem_size_task_index, hipMemcpyHostToDevice));

	hipEventRecord(start);
	dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
	dim3 grid(m/BLOCK_SIZE,t/BLOCK_SIZE,1);
	transposeCoalesced<<<block, grid>>>(d_machines, d_machines_out, d_task_index, d_task_index_out, t, m);
	hipEventRecord(stop);

	hipError_t errSync = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess)
		printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
	if (errAsync != hipSuccess)
		printf("Async kernel error: %s\n", hipGetErrorString(errAsync));

	if (ELAPSED_TIME == 1) {
		hipEventSynchronize(stop);
		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);
		std::cout << milliseconds << "\n";
	}

	hipDeviceSynchronize();

	cudaTest(hipMemcpy(machines, d_machines_out, mem_size_machines, hipMemcpyDeviceToHost));
	cudaTest(hipMemcpy(task_index, d_task_index_out, mem_size_task_index, hipMemcpyDeviceToHost));

	hipFree(d_machines);
	hipFree(d_machines_out);
	hipFree(d_task_index);
	hipFree(d_task_index_out);

	if (ELAPSED_TIME != 1) {
		print(machines, m, t);
		print(task_index, m, t);
	}

	free(machines);
	free(task_index);

	return 0;
}
