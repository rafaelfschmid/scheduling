#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : sorting_segments.cu
 Author      : Rafael Schmid
 Version     :
 Copyright   : Your copyright notice
 Description : Compute sum of reciprocals using STL on CPU and Thrust on GPU
 ============================================================================

 COMPILAR USANDO O SEGUINTE COMANDO:

 nvcc segmented_sort.cu -o segmented_sort -std=c++11 --expt-extended-lambda -I"/home/schmid/Dropbox/Unicamp/workspace/sorting_segments/moderngpu-master/src"

 */

#include <moderngpu/kernel_segsort.hxx>

#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include <hip/hip_runtime.h>

#ifndef ELAPSED_TIME
#define ELAPSED_TIME 0
#endif

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 32
#endif

struct Reduce {
	uint t;
	uint m;
	float value;

	Reduce(uint index_t, uint index_m, float value) {
		this->t = index_t;
		this->m = index_m;
		this->value = value;
	}

	Reduce() {
		this->t = 0;
		this->m = 0;
		this->value = 0.0;
	}
} myobj;

void print(Reduce* vec, uint t, uint m) {
	std::cout << "\n";
	for (uint i = 0; i < t; i++) {
		for (uint j = 0; j < m; j++) {
			std::cout << "t=" << vec[i * m + j].t << " m="
					<< vec[i * m + j].m << " value="
					<< vec[i * m + j].value << "\t||";
		}
		std::cout << "\n";
	}

}

void cudaTest(hipError_t error) {
	if (error != hipSuccess) {
		printf("cuda returned error %s (code %d), line(%d)\n",
				hipGetErrorString(error), error, __LINE__);
		exit (EXIT_FAILURE);
	}
}

template<typename T>
void print(T* vec, uint t, uint m) {
	std::cout << "\n";
	for (uint i = 0; i < t; i++) {
		for (uint j = 0; j < m; j++) {
			std::cout << vec[i * m + j] << " ";
		}
		std::cout << "\n";
	}

}

template<typename T>
void print(T* vec, uint t) {
	std::cout << "\n";
	for (uint i = 0; i < t; i++) {
		std::cout << vec[i] << " ";
	}
	std::cout << "\n";
}

__global__ void calc_completion_times(float* machines, float* completion_times, bool *task_deleted,
		Reduce* completion_aux, int m, int t, float MAX_FLOAT) {

	extern __shared__ Reduce s_comp_times[];

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;


	int tIdrow = threadIdx.y;
	int tIdcol = threadIdx.x;

	int iglobal = row * t + col;
	int ilocal = tIdrow * blockDim.x + tIdcol;

	if(!task_deleted[col]) {
		s_comp_times[ilocal].t = col;
		s_comp_times[ilocal].m = row;
		s_comp_times[ilocal].value = completion_times[row] + machines[iglobal];
	}
	else {
		s_comp_times[ilocal].t = col;
		s_comp_times[ilocal].m = row;
		s_comp_times[ilocal].value = MAX_FLOAT;
	}

	__syncthreads();

	for(int e = blockDim.x/2; e > 0; e/=2)
	{
		if (tIdcol < e) {
			if ((s_comp_times[ilocal + e].value < s_comp_times[ilocal].value)
					|| (s_comp_times[ilocal + e].value == s_comp_times[ilocal].value
							&& s_comp_times[ilocal + e].t < s_comp_times[ilocal].t)) {
				s_comp_times[ilocal].t = s_comp_times[ilocal + e].t;
				s_comp_times[ilocal].m = s_comp_times[ilocal + e].m;
				s_comp_times[ilocal].value = s_comp_times[ilocal + e].value;
			}
		}
		__syncthreads();
	}

	if(tIdcol == 0) {
		for(int e = blockDim.y/2; e > 0; e/=2)
		{
			if (tIdrow < e) {
				if ((s_comp_times[ilocal + e * blockDim.x].value < s_comp_times[ilocal].value)
						|| (s_comp_times[ilocal + e * blockDim.x].value == s_comp_times[ilocal].value
								&& s_comp_times[ilocal + e * blockDim.x].t < s_comp_times[ilocal].t)) {
					s_comp_times[ilocal].t = s_comp_times[ilocal + e * blockDim.x].t;
					s_comp_times[ilocal].m = s_comp_times[ilocal + e * blockDim.x].m;
					s_comp_times[ilocal].value = s_comp_times[ilocal + e * blockDim.x].value;
				}
			}
			__syncthreads();
		}
	}

	if(tIdrow == 0 && tIdcol == 0) {
		iglobal = blockIdx.y * gridDim.x + blockIdx.x;
		completion_aux[iglobal].t = s_comp_times[0].t;
		completion_aux[iglobal].m = s_comp_times[0].m;
		completion_aux[iglobal].value = s_comp_times[0].value;
	}
}

__global__ void reduction_two_dimensional(Reduce* completion_aux, int t) {

	extern __shared__ Reduce s_comp_times[];

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int tIdrow = threadIdx.y;
	int tIdcol = threadIdx.x;

	int iglobal = row * t + col;
	int ilocal = tIdrow * blockDim.x + tIdcol;

	s_comp_times[ilocal].t = completion_aux[iglobal].t;
	s_comp_times[ilocal].m = completion_aux[iglobal].m;
	s_comp_times[ilocal].value = completion_aux[iglobal].value;

	__syncthreads();

	for(int e = blockDim.x/2; e > 0; e/=2)
	{
		if (tIdcol < e) {
			if ((s_comp_times[ilocal + e].value < s_comp_times[ilocal].value)
					|| (s_comp_times[ilocal + e].value == s_comp_times[ilocal].value
							&& s_comp_times[ilocal + e].t < s_comp_times[ilocal].t)) {
				s_comp_times[ilocal].t = s_comp_times[ilocal + e].t;
				s_comp_times[ilocal].m = s_comp_times[ilocal + e].m;
				s_comp_times[ilocal].value = s_comp_times[ilocal + e].value;
			}
		}
		__syncthreads();
	}

	if(tIdcol == 0) {
		for(int e = blockDim.y/2; e > 0; e/=2)
		{
			if (tIdrow < e) {
				if ((s_comp_times[ilocal + e * BLOCK_SIZE].value < s_comp_times[ilocal].value)
						|| (s_comp_times[ilocal + e * BLOCK_SIZE].value == s_comp_times[ilocal].value
								&& s_comp_times[ilocal + e * BLOCK_SIZE].t < s_comp_times[ilocal].t)) {
					s_comp_times[ilocal].t = s_comp_times[ilocal + e * BLOCK_SIZE].t;
					s_comp_times[ilocal].m = s_comp_times[ilocal + e * BLOCK_SIZE].m;
					s_comp_times[ilocal].value = s_comp_times[ilocal + e * BLOCK_SIZE].value;
				}
			}
			__syncthreads();
		}
	}

	if(tIdrow == 0 && tIdcol == 0) {
		iglobal = blockIdx.y * gridDim.x + blockIdx.x;
		completion_aux[iglobal].t = s_comp_times[0].t;
		completion_aux[iglobal].m = s_comp_times[0].m;
		completion_aux[iglobal].value = s_comp_times[0].value;
	}
}

__global__ void reduction(Reduce* d_completion_aux) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int tId = threadIdx.x;

	extern __shared__ Reduce s_comp_times[];

	s_comp_times[tId].t = d_completion_aux[i].t;
	s_comp_times[tId].m = d_completion_aux[i].m;
	s_comp_times[tId].value = d_completion_aux[i].value;

	__syncthreads();

	for(int e = blockDim.x/2; e > 0; e/=2)
	{
		if (tId < e) {
			if ((s_comp_times[tId + e].value < s_comp_times[tId].value)
					|| (s_comp_times[tId + e].value == s_comp_times[tId].value
							&& s_comp_times[tId + e].t < s_comp_times[tId].t)) {
				s_comp_times[tId].t = s_comp_times[tId + e].t;
				s_comp_times[tId].m = s_comp_times[tId + e].m;
				s_comp_times[tId].value = s_comp_times[tId + e].value;
			}
		}
		__syncthreads();
	}

	if(tId == 0) {
		d_completion_aux [blockIdx.x].t = s_comp_times[0].t;
		d_completion_aux[blockIdx.x].m = s_comp_times[0].m;
		d_completion_aux[blockIdx.x].value = s_comp_times[0].value;
	}
}

__global__ void block_reduction(float* completion_times, bool* task_map, bool* task_deleted,
		Reduce* d_completion_aux, int t) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int tId = threadIdx.x;

	extern __shared__ Reduce s_comp_times[];

	s_comp_times[tId].t = d_completion_aux[i].t;
	s_comp_times[tId].m = d_completion_aux[i].m;
	s_comp_times[tId].value = d_completion_aux[i].value;

	__syncthreads();

	for(int e = blockDim.x/2; e > 0; e/=2)
	{
		if (tId < e) {
			if ((s_comp_times[tId + e].value < s_comp_times[tId].value)
					|| (s_comp_times[tId + e].value == s_comp_times[tId].value
							&& s_comp_times[tId + e].t < s_comp_times[tId].t)) {
				s_comp_times[tId].t = s_comp_times[tId + e].t;
				s_comp_times[tId].m = s_comp_times[tId + e].m;
				s_comp_times[tId].value = s_comp_times[tId + e].value;
			}
		}
		__syncthreads();
	}

	if(tId == 0) {
		task_deleted[ s_comp_times[0].t ] = true;
		task_map[ s_comp_times[0].m * t + s_comp_times[0].t ] = true;
		completion_times[ s_comp_times[0].m ] = s_comp_times[0].value;
	}
}


int main(int argc, char** argv) {

	int t, m;

	if (argc < 3) {
			printf("Parameters missing: <number of tasks> <number of machines>\n\n");
			return 0;
	}

	t = atoi(argv[1]);
	m = atoi(argv[2]);

	uint mem_size_machines 			= sizeof(float) * (m * t);
	uint mem_size_completion_times 	= sizeof(float) * (m);
	uint mem_size_task_deleted 		= sizeof(bool) * (t);
	uint mem_size_task_map 			= sizeof(bool) * (m * t);

	int dimCol = (t-1)/BLOCK_SIZE + 1;
	int dimRow = (m-1)/BLOCK_SIZE + 1;

	uint mem_size_completion_aux	= sizeof(Reduce) * (dimCol * dimRow);

	float *machines				= (float *) malloc(mem_size_machines);
	float *completion_times 	= (float *) malloc(mem_size_completion_times);
	bool *task_deleted			= (bool  *) malloc(mem_size_task_deleted);
	bool *task_map 				= (bool  *) malloc(mem_size_task_map);

	float aux;
	for (int i = 0; i < t; i++) {
		for (int j = 0; j < m; j++) {
			int a = scanf("%f", &aux);

			machines[j * t + i] = aux;
			task_map[j * t + i] = false;
			completion_times[j] = 0;
		}
		task_deleted[i] = false;
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	float *d_machines, *d_completion_times;
	bool *d_task_deleted, *d_task_map;
	Reduce *d_completion_aux;
	float MAX_FLOAT = std::numeric_limits<float>::max();

	cudaTest(hipMalloc((void **) &d_machines, mem_size_machines));
	cudaTest(hipMalloc((void **) &d_completion_times, mem_size_completion_times));
	cudaTest(hipMalloc((void **) &d_task_deleted, mem_size_task_deleted));
	cudaTest(hipMalloc((void **) &d_task_map, mem_size_task_map));

	cudaTest(hipMalloc((void **) &d_completion_aux, mem_size_completion_aux));

	// copy host memory to device
	cudaTest(hipMemcpy(d_machines, machines, mem_size_machines, hipMemcpyHostToDevice));
	cudaTest(hipMemcpy(d_completion_times, completion_times, mem_size_completion_times, hipMemcpyHostToDevice));
	cudaTest(hipMemcpy(d_task_deleted, task_deleted, mem_size_task_deleted, hipMemcpyHostToDevice));
	cudaTest(hipMemcpy(d_task_map, task_map, mem_size_task_map, hipMemcpyHostToDevice));

	hipEventRecord(start);
	for(int k = 0; k < t; k++) {
		dimCol = (t-1)/BLOCK_SIZE + 1;
		dimRow = (m-1)/BLOCK_SIZE + 1;

		dim3 dimB1(BLOCK_SIZE, BLOCK_SIZE);
		dim3 dimG1(dimCol, dimRow);
		calc_completion_times<<<dimG1, dimB1, BLOCK_SIZE * BLOCK_SIZE * sizeof(Reduce) >>>
				(d_machines, d_completion_times, d_task_deleted, d_completion_aux, m, t, MAX_FLOAT);



/*		Reduce *completion_aux 	= (Reduce *) malloc(mem_size_completion_aux);
		cudaTest(hipMemcpy(completion_aux, d_completion_aux, mem_size_completion_aux, hipMemcpyDeviceToHost));
		print(completion_aux, dimRow, dimCol);*/

/*		dimCol = (dimCol-1)/BLOCK_SIZE + 1;
		dimRow = (dimRow-1)/BLOCK_SIZE + 1;

		for( ; dimRow > BLOCK_SIZE; dimRow/=BLOCK_SIZE) {
			dim3 dimG2(dimCol, dimRow);

			reduction_two_dimensional<<<dimG2, dimB1, BLOCK_SIZE * BLOCK_SIZE * sizeof(Reduce) >>>	(d_completion_aux, t);

			dimCol /= BLOCK_SIZE;
		}

		dim3 dimB3(BLOCK_SIZE, dimRow);
		dim3 dimG3(dimCol);
		reduction_two_dimensional<<<dimG3, dimB3, dimRow * BLOCK_SIZE * sizeof(Reduce) >>> (d_completion_aux, t);

		cudaTest(hipMemcpy(completion_aux, d_completion_aux, mem_size_completion_aux, hipMemcpyDeviceToHost));
		print(completion_aux, dimRow, dimCol);

		for( ; dimCol > BLOCK_SIZE; dimCol/=BLOCK_SIZE) {
			dim3 dimB4(BLOCK_SIZE);
			dim3 dimG4(dimCol);

			reduction<<<dimG4, dimB4, BLOCK_SIZE * sizeof(Reduce) >>> (d_completion_aux);
		}

		dim3 dimB5(dimCol);
		dim3 dimG5(1);
		block_reduction<<<dimG5, dimB5, dimCol * sizeof(Reduce) >>> (d_completion_times, d_task_map, d_task_deleted,
				d_completion_aux, t);*/
	}
	hipEventRecord(stop);

	hipError_t errSync = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess)
		printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
	if (errAsync != hipSuccess)
		printf("Async kernel error: %s\n", hipGetErrorString(errAsync));

	if (ELAPSED_TIME == 1) {
		hipEventSynchronize(stop);
		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);
		std::cout << milliseconds << "\n";
	}

	hipDeviceSynchronize();

	cudaTest(hipMemcpy(completion_times, d_completion_times, mem_size_completion_times, hipMemcpyDeviceToHost));
	cudaTest(hipMemcpy(task_map, d_task_map, mem_size_task_map, hipMemcpyDeviceToHost));

	hipFree(d_machines);
	hipFree(d_completion_times);
	hipFree(d_task_map);
	hipFree(d_task_deleted);

	hipFree(d_completion_aux);

	if (ELAPSED_TIME != 1) {
		//print(machines, m, t);
		//print(task_index, m, t);
		print(completion_times, m);
	}

	free(task_deleted);
	free(task_map);
	free(machines);
	free(completion_times);

	return 0;
}

/*
 *
 *
__global__ void block_reduction_two_dimensional(float* completion_times,
		bool* task_map, bool* task_deleted, Reduce* completion_aux, int t) {

	extern __shared__ Reduce s_comp_times[];

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int tIdrow = threadIdx.y;
	int tIdcol = threadIdx.x;

	int iglobal = row * t + col;
	int ilocal = tIdrow * blockDim.x + tIdcol;

	s_comp_times[ilocal].t = completion_aux[iglobal].t;
	s_comp_times[ilocal].m = completion_aux[iglobal].m;
	s_comp_times[ilocal].value = completion_aux[iglobal].value;

	__syncthreads();

	for(int e = blockDim.x/2; e > 0; e/=2)
	{
		if (tIdcol < e) {
			if ((s_comp_times[ilocal + e].value < s_comp_times[ilocal].value)
					|| (s_comp_times[ilocal + e].value == s_comp_times[ilocal].value
							&& s_comp_times[ilocal + e].t < s_comp_times[ilocal].t)) {
				s_comp_times[ilocal].t = s_comp_times[ilocal + e].t;
				s_comp_times[ilocal].m = s_comp_times[ilocal + e].m;
				s_comp_times[ilocal].value = s_comp_times[ilocal + e].value;
			}
		}
		__syncthreads();
	}

	if(tIdcol == 0) {
		for(int e = blockDim.y/2; e > 0; e/=2)
		{
			if (tIdrow < e) {
				if ((s_comp_times[ilocal + e * BLOCK_SIZE].value < s_comp_times[ilocal].value)
						|| (s_comp_times[ilocal + e * BLOCK_SIZE].value == s_comp_times[ilocal].value
								&& s_comp_times[ilocal + e * BLOCK_SIZE].t < s_comp_times[ilocal].t)) {
					s_comp_times[ilocal].t = s_comp_times[ilocal + e * BLOCK_SIZE].t;
					s_comp_times[ilocal].m = s_comp_times[ilocal + e * BLOCK_SIZE].m;
					s_comp_times[ilocal].value = s_comp_times[ilocal + e * BLOCK_SIZE].value;
				}
			}
			__syncthreads();
		}
	}

	if(tIdrow == 0 && tIdcol == 0) {
		task_deleted[ s_comp_times[0].t ] = true;
		task_map[ s_comp_times[0].m * t + s_comp_times[0].t ] = true;
		completion_times[ s_comp_times[0].m ] = s_comp_times[0].value;
	}
}
__global__ void calc_completion_times(float* machines, float* completion_times, bool *task_deleted,
		Reduce* completion_aux, int m, int t, float MAX_FLOAT) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if(!task_deleted[i]) {
		for (int j = 0; j < m; j++) {
			completion_aux[j * t + i].t = i;
			completion_aux[j * t + i].m = j;
			completion_aux[j * t + i].value = completion_times[j] + machines[j * t + i];
		}
	}
	else {
		for (int j = 0; j < m; j++) {
			completion_aux[j * t + i].t = i;
			completion_aux[j * t + i].m = j;
			completion_aux[j * t + i].value = MAX_FLOAT;
		}
	}
}

__global__ void reduction(Reduce* d_completion_aux) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int tId = threadIdx.x;

	extern __shared__ Reduce s_comp_times[];

	s_comp_times[tId].t = d_completion_aux[i].t;
	s_comp_times[tId].m = d_completion_aux[i].m;
	s_comp_times[tId].value = d_completion_aux[i].value;

	__syncthreads();

	for(int e = blockDim.x/2; e > 0; e/=2)
	{
		if (tId < e) {
			if ((s_comp_times[tId + e].value < s_comp_times[tId].value)
					|| (s_comp_times[tId + e].value == s_comp_times[tId].value
							&& s_comp_times[tId + e].t < s_comp_times[tId].t)) {
				s_comp_times[tId].t = s_comp_times[tId + e].t;
				s_comp_times[tId].m = s_comp_times[tId + e].m;
				s_comp_times[tId].value = s_comp_times[tId + e].value;
			}
		}
		__syncthreads();
	}

	if(tId == 0) {
		d_completion_aux [blockIdx.x].t = s_comp_times[0].t;
		d_completion_aux[blockIdx.x].m = s_comp_times[0].m;
		d_completion_aux[blockIdx.x].value = s_comp_times[0].value;
	}
}

__global__ void block_reduction(float* completion_times, bool* task_map, bool* task_deleted,
		Reduce* d_completion_aux, int t) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int tId = threadIdx.x;

	extern __shared__ Reduce s_comp_times[];

	s_comp_times[tId].t = d_completion_aux[i].t;
	s_comp_times[tId].m = d_completion_aux[i].m;
	s_comp_times[tId].value = d_completion_aux[i].value;

	__syncthreads();

	for(int e = blockDim.x/2; e > 0; e/=2)
	{
		if (tId < e) {
			if ((s_comp_times[tId + e].value < s_comp_times[tId].value)
					|| (s_comp_times[tId + e].value == s_comp_times[tId].value
							&& s_comp_times[tId + e].t < s_comp_times[tId].t)) {
				s_comp_times[tId].t = s_comp_times[tId + e].t;
				s_comp_times[tId].m = s_comp_times[tId + e].m;
				s_comp_times[tId].value = s_comp_times[tId + e].value;
			}
		}
		__syncthreads();
	}

	if(tId == 0) {
		task_deleted[ s_comp_times[0].t ] = true;
		task_map[ s_comp_times[0].m * t + s_comp_times[0].t ] = true;
		completion_times[ s_comp_times[0].m ] = s_comp_times[0].value;
	}
}

int main(int argc, char** argv) {

	int t, m;

	if (argc < 3) {
			printf("Parameters missing: <number of tasks> <number of machines>\n\n");
			return 0;
	}

	t = atoi(argv[1]);
	m = atoi(argv[2]);

	uint mem_size_machines 			= sizeof(float) * (m * t);
	uint mem_size_completion_times 	= sizeof(float) * (m);
	uint mem_size_task_deleted 		= sizeof(bool) * (t);
	uint mem_size_task_map 			= sizeof(bool) * (m * t);
	uint mem_size_completion_aux	= sizeof(Reduce) * (m * t);

	float *machines				= (float *) malloc(mem_size_machines);
	float *completion_times 	= (float *) malloc(mem_size_completion_times);
	bool *task_deleted			= (bool  *) malloc(mem_size_task_deleted);
	bool *task_map 				= (bool  *) malloc(mem_size_task_map);

	float aux;
	for (int i = 0; i < t; i++) {
		for (int j = 0; j < m; j++) {
			int a = scanf("%f", &aux);

			machines[j * t + i] = aux;
			task_map[j * t + i] = false;
			completion_times[j] = 0;
		}
		task_deleted[i] = false;
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	float *d_machines, *d_completion_times;
	bool *d_task_deleted, *d_task_map;
	Reduce *d_completion_aux;
	float MAX_FLOAT = std::numeric_limits<float>::max();

	cudaTest(hipMalloc((void **) &d_machines, mem_size_machines));
	cudaTest(hipMalloc((void **) &d_completion_times, mem_size_completion_times));
	cudaTest(hipMalloc((void **) &d_task_deleted, mem_size_task_deleted));
	cudaTest(hipMalloc((void **) &d_task_map, mem_size_task_map));

	cudaTest(hipMalloc((void **) &d_completion_aux, mem_size_completion_aux));

	// copy host memory to device
	cudaTest(hipMemcpy(d_machines, machines, mem_size_machines, hipMemcpyHostToDevice));
	cudaTest(hipMemcpy(d_completion_times, completion_times, mem_size_completion_times, hipMemcpyHostToDevice));
	cudaTest(hipMemcpy(d_task_deleted, task_deleted, mem_size_task_deleted, hipMemcpyHostToDevice));
	cudaTest(hipMemcpy(d_task_map, task_map, mem_size_task_map, hipMemcpyHostToDevice));


	hipEventRecord(start);
	for(int k = 0; k < t; k++) {
		int dimG = t * m;

		int dim = t/BLOCK_SIZE;
		dim3 dimBlock(BLOCK_SIZE);
		dim3 dimGrid(dim);
		calc_completion_times<<<dimGrid, dimBlock>>>
				(d_machines, d_completion_times, d_task_deleted, d_completion_aux, m, t, MAX_FLOAT);

		for( ; dimG > BLOCK_SIZE; dimG/=BLOCK_SIZE) {
			dim3 block(BLOCK_SIZE);
			dim3 grid_b(dimG/BLOCK_SIZE);
			reduction<<<grid_b, block, BLOCK_SIZE * sizeof(Reduce) >>>
				(d_completion_aux);
		}

		dim3 block(dimG);
		dim3 grid_b(1);
		block_reduction<<<grid_b, block, dimG * sizeof(Reduce) >>> (d_completion_times, d_task_map, d_task_deleted,
				d_completion_aux, t);
	}
	hipEventRecord(stop);

	hipError_t errSync = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess)
		printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
	if (errAsync != hipSuccess)
		printf("Async kernel error: %s\n", hipGetErrorString(errAsync));

	if (ELAPSED_TIME == 1) {
		hipEventSynchronize(stop);
		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);
		std::cout << milliseconds << "\n";
	}

	hipDeviceSynchronize();

	cudaTest(hipMemcpy(completion_times, d_completion_times, mem_size_completion_times, hipMemcpyDeviceToHost));
	cudaTest(hipMemcpy(task_map, d_task_map, mem_size_task_map, hipMemcpyDeviceToHost));

	hipFree(d_machines);
	hipFree(d_completion_times);
	hipFree(d_task_map);
	hipFree(d_task_deleted);

	hipFree(d_completion_aux);

	if (ELAPSED_TIME != 1) {
		//print(machines, m, t);
		//print(task_index, m, t);
		print(completion_times, m);
	}

	free(task_deleted);
	free(task_map);
	free(machines);
	free(completion_times);

	return 0;
}
 *
 */
